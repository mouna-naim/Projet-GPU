#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>
#include <omp.h>
#include <hip/hip_runtime.h>


#define ELEMENT_TYPE float
#define THREAD_PER_BLOCK 265 

#define DEFAULT_ARRAY_LEN 8000 //10
#define DEFAULT_NB_BINS 6000
#define DEFAULT_LOWER_BOUND 0.0
#define DEFAULT_UPPER_BOUND 10.0
#define DEFAULT_NB_REPEAT 10

#define MAX_DISPLAY_COLUMNS 10
#define MAX_DISPLAY_ROWS 20

struct s_settings
{
        int array_len;
        int nb_bins;
        double lower_bound;
        double upper_bound;
        int nb_repeat;
        int enable_output;
        int enable_verbose;
};

#define PRINT_ERROR(MSG)                                                    \
        do                                                                  \
        {                                                                   \
                fprintf(stderr, "%s:%d - %s\n", __FILE__, __LINE__, (MSG)); \
                exit(EXIT_FAILURE);                                         \
        } while (0)

#define IO_CHECK(OP, RET)                   \
        do                                  \
        {                                   \
                if ((RET) < 0)              \
                {                           \
                        perror((OP));       \
                        exit(EXIT_FAILURE); \
                }                           \
        } while (0)

static void usage(void)
{
        fprintf(stderr, "usage: histogram [OPTIONS...]\n");
        fprintf(stderr, "    --array-len  ARRAY_LENGTH\n");
        fprintf(stderr, "    --nb-bins  NB_BINS\n");
        fprintf(stderr, "    --lower-bound  LOWER_BOUND\n");
        fprintf(stderr, "    --higher-bound  HIGHER_BOUND\n");
        fprintf(stderr, "    --nb-repeat NB_REPEAT\n");
        fprintf(stderr, "    --output\n");
        fprintf(stderr, "    --verbose\n");
        fprintf(stderr, "\n");
        exit(EXIT_FAILURE);
}
static void init_settings(struct s_settings **pp_settings)
{
        assert(*pp_settings == NULL);

        struct s_settings *p_settings = (struct s_settings *)calloc(1, sizeof(struct s_settings));        
        if (p_settings == NULL)
        {
                PRINT_ERROR("memory allocation failed");
                exit(EXIT_FAILURE);
        }
        p_settings->array_len = DEFAULT_ARRAY_LEN;
        p_settings->nb_bins = DEFAULT_NB_BINS;
        p_settings->lower_bound = DEFAULT_LOWER_BOUND;
        p_settings->upper_bound = DEFAULT_UPPER_BOUND;
        p_settings->nb_repeat = DEFAULT_NB_REPEAT;
        p_settings->enable_verbose = 0;
        p_settings->enable_output = 0;
        *pp_settings = p_settings;
}

static void parse_cmd_line(int argc, char *argv[], struct s_settings *p_settings)
{
        int i = 1;
        while (i < argc)
        {
                if (strcmp(argv[i], "--array-len") == 0)
                {
                        i++;
                        if (i >= argc)
                        {
                                usage();
                        }
                        int value = atoi(argv[i]);
                        if (value < 1)
                        {
                                fprintf(stderr, "invalid ARRAY_LENGTH argument\n");
                                exit(EXIT_FAILURE);
                        }
                        p_settings->array_len = value;
                }
                else if (strcmp(argv[i], "--nb-bins") == 0)
                {
                        i++;
                        if (i >= argc)
                        {
                                usage();
                        }
                        int value = atoi(argv[i]);
                        if (value < 1)
                        {
                                fprintf(stderr, "invalid NB_BINS argument\n");
                                exit(EXIT_FAILURE);
                        }
                        p_settings->nb_bins = value;
                }
                else if (strcmp(argv[i], "--lower-bound") == 0)
                {
                        i++;
                        if (i >= argc)
                        {
                                usage();
                        }
                        int value = atof(argv[i]);
                        int class_value =fpclassify(value);
                        if ((class_value != FP_NORMAL) && (class_value != FP_ZERO))
                        {
                                fprintf(stderr, "invalid LOWER_BOUND argument\n");
                                exit(EXIT_FAILURE);
                        }
                        p_settings->lower_bound = value;
                }
                else if (strcmp(argv[i], "--upper-bound") == 0)
                {
                        i++;
                        if (i >= argc)
                        {
                                usage();
                        }
                        double value = atof(argv[i]);
                        int class_value = fpclassify(value);
                        if ((class_value != FP_NORMAL) && (class_value != FP_ZERO))
                        {
                                fprintf(stderr, "invalid UPPER_BOUND argument\n");
                                exit(EXIT_FAILURE);
                        }
                        p_settings->upper_bound = value;
                }
                else if (strcmp(argv[i], "--nb-repeat") == 0)
                {
                        i++;
                        if (i >= argc)
                        {
                                usage();
                        }
                        int value = atoi(argv[i]);
                        if (value < 1)
                        {
                                fprintf(stderr, "invalid NB_REPEAT argument\n");
                                exit(EXIT_FAILURE);
                        }
                        p_settings->nb_repeat = value;
                }
                else if (strcmp(argv[i], "--output") == 0)
                {
                        p_settings->enable_output = 1;
                }
                else if (strcmp(argv[i], "--verbose") == 0)
                {
                        p_settings->enable_verbose = 1;
                }
                else
                {
                        usage();
                }

                i++;
        }

        if (p_settings->upper_bound <= p_settings->lower_bound)
        {
                fprintf(stderr, "invalid histogram bounds\n");
                exit(EXIT_FAILURE);
        }

        if (p_settings->enable_output)
        {
                p_settings->nb_repeat = 1;
        }
}

static void delete_settings(struct s_settings **pp_settings)
{
        assert(*pp_settings != NULL);
        free(*pp_settings);
        pp_settings = NULL;
}

static void allocate_array(ELEMENT_TYPE **p_array, struct s_settings *p_settings)
{
        assert(*p_array == NULL);
        ELEMENT_TYPE *array = (float *)calloc(p_settings->array_len, sizeof(*array));
        if (array == NULL)
        {
                PRINT_ERROR("memory allocation failed");
        }
        *p_array = array;
}

static void delete_array(ELEMENT_TYPE **p_array)
{
        assert(*p_array != NULL);
        free(*p_array);
        p_array = NULL;
}

static void init_array_random(ELEMENT_TYPE *array, struct s_settings *p_settings)
{
        const ELEMENT_TYPE offset = p_settings->lower_bound;
        const ELEMENT_TYPE scale = p_settings->upper_bound - p_settings->lower_bound;

        int i;
        for (i = 0; i < p_settings->array_len; i++)
        {
                ELEMENT_TYPE value = scale * ((ELEMENT_TYPE)rand()) / (1.0 + (ELEMENT_TYPE)(RAND_MAX)) + offset;
                array[i] = value;
        }
}

static void print_array(const ELEMENT_TYPE *array, struct s_settings *p_settings)
{
        printf("[");
        int j = 0;
        int i;
        for (i = 0; i < p_settings->array_len; i++)
        {
                if (i > 0)
                {
                        printf(",");
                        if ((i % MAX_DISPLAY_COLUMNS == 0))
                        {
                                printf("\n");
                                printf(" ");
                                j++;

                                if (j >= MAX_DISPLAY_ROWS)
                                {
                                        printf("  ...\n");
                                        break;
                                }
                        }
                }
                printf(" %8.3lg", array[i]);
        }
        printf(" ]");
}

static void write_array_to_file(FILE *file, const ELEMENT_TYPE *array, struct s_settings *p_settings)
{
        int i;
        int ret;

        for (i = 0; i < p_settings->array_len; i++)
        {
                ret = fprintf(file, "%lf\n", array[i]);
                IO_CHECK("fprintf", ret);
        }
}

static void allocate_histogram(int **p_histogram, struct s_settings *p_settings)
{
        assert(*p_histogram == NULL);
        int *histogram =(int *)calloc(p_settings->nb_bins, sizeof(*histogram));
        if (histogram == NULL)
        {
                PRINT_ERROR("memory allocation failed");
        }
        *p_histogram = histogram;
}

static void delete_histogram(int **p_histogram)
{
        assert(*p_histogram != NULL);
        free(*p_histogram);
        p_histogram = NULL;
}

static void print_histogram(const int *histogram, struct s_settings *p_settings)
{
        const ELEMENT_TYPE offset = p_settings->lower_bound;
        const ELEMENT_TYPE scale = p_settings->upper_bound - p_settings->lower_bound;

        printf("<\n");
        int i;
        for (i = 0; i < p_settings->nb_bins; i++)
        {
                ELEMENT_TYPE lower = offset + i * scale / p_settings->nb_bins;
                ELEMENT_TYPE upper = offset + (i + 1) * scale / p_settings->nb_bins;

                printf(" [ %8.2lg ... %8.2lg [ :  %d\n", lower, upper, histogram[i]);
        }
        printf(">");
}

static void write_bins_to_file(FILE *file, struct s_settings *p_settings)
{
        int i;
        int ret;

        const ELEMENT_TYPE offset = p_settings->lower_bound;
        const ELEMENT_TYPE scale = p_settings->upper_bound - p_settings->lower_bound;

        ret = fprintf(file, "%lf\n", offset);
        IO_CHECK("fprintf", ret);
        for (i = 0; i < p_settings->nb_bins; i++)
        {
                ELEMENT_TYPE bound = offset + (i + 1) * scale / p_settings->nb_bins;
                ret = fprintf(file, "%lf\n", bound);
                IO_CHECK("fprintf", ret);
        }
}

static void write_histogram_to_file(FILE *file, const int *histogram, struct s_settings *p_settings)
{
        int i;
        int ret;

        for (i = 0; i < p_settings->nb_bins; i++)
        {
                ret = fprintf(file, "%d\n", histogram[i]);
                IO_CHECK("fprintf", ret);
        }
}

static void print_settings_csv_header(void)
{
        printf("array_len,nb_bins,nb_repeat");
}

static void print_settings_csv(struct s_settings *p_settings)
{
        printf("%d,%d,%d", p_settings->array_len, p_settings->nb_bins, p_settings->nb_repeat);
}

static void print_results_csv_header(void)
{
        printf("rep,timing,check_status");
}

static void print_results_csv(int rep, double timing_in_seconds, int check_status)
{
        printf("%d,%le,%d", rep, timing_in_seconds, check_status);
}

static void print_csv_header(void)
{
        print_settings_csv_header();
        printf(",");
        print_results_csv_header();
        printf("\n");
}

static void naive_compute_histogram(const ELEMENT_TYPE *array, int *histogram, struct s_settings *p_settings)
{
        //Intialiser l'histogramme à zéro
        memset(histogram, 0, p_settings->nb_bins * sizeof(*histogram));

        ELEMENT_TYPE *bounds = NULL;
        bounds = (float *)malloc((p_settings->nb_bins + 1) * sizeof(*bounds));
        if (bounds == NULL)
        {
                PRINT_ERROR("memory allocation failed");
        }

        {
                const ELEMENT_TYPE offset = p_settings->lower_bound;
                const ELEMENT_TYPE scale = p_settings->upper_bound - p_settings->lower_bound;

                bounds[0] = offset;

                //On peut paralléliser avec Openmp
                int j;
                //#pragma omp parallel for
                for (j = 0; j < p_settings->nb_bins; j++)
                {
                        bounds[j + 1] = offset + (j + 1) * scale / p_settings->nb_bins;
                }
        }

        
        int i;
        for (i = 0; i < p_settings->array_len; i++)
        {
                ELEMENT_TYPE value = array[i];

                int j;
                for (j = 0; j < p_settings->nb_bins; j++)
                {
                        if (value >= bounds[j] && value < bounds[j + 1])
                        {
                                histogram[j]++;
                                break;
                        }
                }
        }

        free(bounds);
}

//Appelée depuis le CPU mais executée sur le GPU__Déclaration du kernel Cuda
__global__ void compute_histogram_kernel(const ELEMENT_TYPE *array, int *histogram, const ELEMENT_TYPE *bounds, int array_len, int nb_bins) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < array_len) {

        ELEMENT_TYPE value = array[i];

        for (int j = 0; j < nb_bins; j++) {

            if (value >= bounds[j] && value < bounds[j + 1]) {

                atomicAdd(&histogram[j], 1);

                break;
            }
        }
    }
}

static void cuda_compute_histogram(const ELEMENT_TYPE *array, int *histogram, struct s_settings *p_settings) 
{
    //Intialiser l'histogramme à zéro
        memset(histogram, 0, p_settings->nb_bins * sizeof(*histogram));

        ELEMENT_TYPE *bounds = NULL;
        bounds = (float *)malloc((p_settings->nb_bins + 1) * sizeof(*bounds));
        if (bounds == NULL)
        {
                PRINT_ERROR("memory allocation failed");
        }

        {
                const ELEMENT_TYPE offset = p_settings->lower_bound;
                const ELEMENT_TYPE scale = p_settings->upper_bound - p_settings->lower_bound;

                bounds[0] = offset;

                //On peut paralléliser avec Openmp
                int j;
                //#pragma omp parallel for
                for (j = 0; j < p_settings->nb_bins; j++)
                {
                        bounds[j + 1] = offset + (j + 1) * scale / p_settings->nb_bins;
                }
        }

        ELEMENT_TYPE *gpu_array, *gpu_bounds;
        int *gpu_histogram;

        //Allouer la mémoire sur le GPU
        hipMalloc((void **)&gpu_array, p_settings ->array_len * sizeof(ELEMENT_TYPE));
        hipMalloc((void**)&gpu_bounds, (p_settings->nb_bins + 1) * sizeof(ELEMENT_TYPE));
        hipMalloc((void**)&gpu_histogram, p_settings->nb_bins * sizeof(int));

        //Copier les données depuis le CPU vers le GPU
        hipMemcpy(gpu_array, array, p_settings->array_len * sizeof(ELEMENT_TYPE), hipMemcpyHostToDevice);
        hipMemcpy(gpu_bounds, bounds, (p_settings->nb_bins + 1) * sizeof(ELEMENT_TYPE), hipMemcpyHostToDevice);

        int num_blocks = (p_settings->array_len + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

        //Appeler le kernel cuda
        compute_histogram_kernel<<<num_blocks, THREAD_PER_BLOCK>>>(gpu_array, gpu_histogram, gpu_bounds, p_settings->array_len, p_settings->nb_bins);

        //Copierles résultats depuis le GPU vers le CPU
        hipMemcpy(histogram, gpu_histogram, p_settings->nb_bins * sizeof(int), hipMemcpyDeviceToHost);

        //Libérer la mémoire allouée sur le GPU
        hipFree(gpu_array);
        hipFree(gpu_bounds);
        hipFree(gpu_histogram);

        // Libérer la mémoire allouée sur le CPU
        free(bounds);

}


static void run(const ELEMENT_TYPE *array, int *run_histogram, struct s_settings *p_settings)
{
        //mp_compute_histogram(array, run_histogram, p_settings);
        //naive_compute_histogram(array, run_histogram, p_settings);
        cuda_compute_histogram(array, run_histogram, p_settings);
        if (p_settings->enable_output)
        {
                FILE *file = fopen("run_histogram.csv", "w");
                if (file == NULL)
                {
                        perror("fopen");
                        exit(EXIT_FAILURE);
                }
                write_histogram_to_file(file, run_histogram, p_settings);
                fclose(file);
        }

        if (p_settings->enable_verbose)
        {
                printf("run histogram:\n");
                print_histogram(run_histogram, p_settings);
                printf("\n\n");
        }
}

static int check(const ELEMENT_TYPE *array, int *check_histogram, const int *run_histogram, struct s_settings *p_settings)
{
        naive_compute_histogram(array, check_histogram, p_settings);

        if (p_settings->enable_output)
        {
                FILE *file = fopen("check_histogram.csv", "w");
                if (file == NULL)
                {
                        perror("fopen");
                        exit(EXIT_FAILURE);
                }
                write_histogram_to_file(file, check_histogram, p_settings);
                fclose(file);
        }

        if (p_settings->enable_verbose)
        {
                printf("check histogram:\n");
                print_histogram(check_histogram, p_settings);
                printf("\n\n");
        }

        int check = 0;
        int i;
        for (i = 0; i < p_settings->nb_bins; i++)
        {
                if (run_histogram[i] != check_histogram[i])
                {
                        fprintf(stderr, "check failed [bin: %d]: run = %d, check = %d\n", i,
                                run_histogram[i], check_histogram[i]);
                        check = 1;
                }
        }
        return check;
}

int main(int argc, char *argv[])
{
        struct s_settings *p_settings = NULL;

        init_settings(&p_settings);
        parse_cmd_line(argc, argv, p_settings);

        ELEMENT_TYPE *array = NULL;
        allocate_array(&array, p_settings);

        int *histogram = NULL;
        allocate_histogram(&histogram, p_settings);

        int *check_histogram = NULL;
        allocate_histogram(&check_histogram, p_settings);

        {
                if (!p_settings->enable_verbose)
                {
                        print_csv_header();
                }

                if (p_settings->enable_output)
                {
                        FILE *file = fopen("bins.csv", "w");
                        if (file == NULL)
                        {
                                perror("fopen");
                                exit(EXIT_FAILURE);
                        }
                        write_bins_to_file(file, p_settings);
                        fclose(file);
                }

                int rep;
                for (rep = 0; rep < p_settings->nb_repeat; rep++)
                {
                        if (p_settings->enable_verbose)
                        {
                                printf("repeat %d\n", rep);
                        }

                        init_array_random(array, p_settings);

                        if (p_settings->enable_output)
                        {
                                FILE *file = fopen("array.csv", "w");
                                if (file == NULL)
                                {
                                        perror("fopen");
                                        exit(EXIT_FAILURE);
                                }
                                write_array_to_file(file, array, p_settings);
                                fclose(file);
                        }

                        if (p_settings->enable_verbose)
                        {
                                printf("array:\n");
                                print_array(array, p_settings);
                                printf("\n\n");
                        }

                        struct timespec timing_start, timing_end;
                        clock_gettime(CLOCK_MONOTONIC, &timing_start);
                        run(array, histogram, p_settings);
                        clock_gettime(CLOCK_MONOTONIC, &timing_end);
                        double timing_in_seconds = (timing_end.tv_sec - timing_start.tv_sec) + 1.0e-9 * (timing_end.tv_nsec - timing_start.tv_nsec);

                        int check_status = check(array, check_histogram, histogram, p_settings);

                        if (p_settings->enable_verbose)
                        {
                                print_csv_header();
                        }
                        print_settings_csv(p_settings);
                        printf(",");
                        print_results_csv(rep, timing_in_seconds, check_status);
                        printf("\n");
                }
        }

        delete_histogram(&check_histogram);
        delete_histogram(&histogram);

        delete_array(&array);
        delete_settings(&p_settings);

        return 0;
}
